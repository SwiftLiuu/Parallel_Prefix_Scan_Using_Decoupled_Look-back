#include "hip/hip_runtime.h"
#include "implementation.h"

#include "stdio.h"

void printSubmissionInfo()
{
    // This will be published in the leaderboard on piazza
    // Please modify this field with something interesting
    char nick_name[] = "Decoupled-Look-Back-Team";

    // Please fill in your information (for marking purposes only)
    char student_first_name[] = "Xiangyu";
    char student_last_name[] = "Liu";
    char student_student_number[] = "1006743179";

    // Printing out team information
    printf("*******************************************************************************************************\n");
    printf("Submission Information:\n");
    printf("\tnick_name: %s\n", nick_name);
    printf("\tstudent_first_name: %s\n", student_first_name);
    printf("\tstudent_last_name: %s\n", student_last_name);
    printf("\tstudent_student_number: %s\n", student_student_number);
}

/**
 * Implement your CUDA inclusive scan here. Feel free to add helper functions, kernels or allocate temporary memory.
 * However, you must not modify other files. CAUTION: make sure you synchronize your kernels properly and free all
 * allocated memory.
 *
 * @param d_input: input array on device
 * @param d_output: output array on device
 * @param size: number of elements in the input array
 */





__global__ void koggeStoneDecoupledLookbackKernel(const int32_t *d_input, int32_t *d_output, volatile uint64_t *d_aggregates, size_t size, size_t block_workload) {
    extern __shared__ int32_t shared_mem[]; 
    int32_t *s_data = shared_mem;           

    int tid = threadIdx.x;                  
    int blockStart = blockIdx.x * block_workload; 
    int end = min(blockStart + block_workload, size); // End index for this block

    // First Offset Calculation Directly Using Global Memory (offset = 1)
    for (int i = blockStart + tid; i < end; i += blockDim.x) {
        if (i > blockStart) { 
            s_data[i - blockStart] = d_input[i] + d_input[i - 1];
        } else {
            // First element has no previous element
            s_data[i - blockStart] = d_input[i];
        }
    }
    __syncthreads();

     // Perform Remaining Offsets Using Shared Memory (offset = 2, 4, ...)
    for (int offset = 2; offset < block_workload; offset *= 2) {
        // Calculate the lower bound for valid iterations in this offset
       int lower_bound = 0 - (blockDim.x - (block_workload - offset) % blockDim.x);

       // Reverse iteration ensures updates are not read by other threads
       for (int i = block_workload - 1 - tid; i - offset >= lower_bound; i -= blockDim.x) {
           int32_t local_temp = 0;
   
           if (i >= 0 && i - offset >= 0) {
               local_temp = s_data[i - offset];
           }
       
   
           // Synchronize threads to ensure all threads have finished reading
           __syncthreads();
   
           if (i >= 0) { 
               s_data[i] += local_temp;
           }
   
       }
   
       // Synchronize threads after completing all updates for the current offset
       __syncthreads();
   }



    // Thread 0 sets the aggregate and flag
    if (tid == 0) {
        int32_t block_aggregate = s_data[block_workload - 1]; 
        uint64_t encoded_value = ((uint64_t)block_aggregate << 2); // Shift aggregate to the left by 2 to spare spaces for the flag

        if (blockIdx.x == 0) {
            d_aggregates[blockIdx.x] = encoded_value | 0b10; // Set flag to 'P' (pre-aggregate available)
        } else {
            d_aggregates[blockIdx.x] = encoded_value | 0b01; // Set flag to 'A' (aggregate available)
        }
        __threadfence();
    }
    



    // Decoupled Lookback for Pre-Aggregate (Only for Non-Block 0)
    uint64_t pre_aggregate = 0;
    if (blockIdx.x > 0 && tid == 0) {
        for (int i = blockIdx.x - 1; i >= 0; i--) {
            uint64_t value;
            do {
                value = d_aggregates[i];
            } while ((value & 0b11) == 0); // Poll until flag is not 'X'

            uint64_t aggregate = value >> 2;
            uint64_t flag = value & 0b11;

            if (flag == 0b01) { // 'A' (aggregate available)
                pre_aggregate += aggregate;
            } else if (flag == 0b10) { // 'P' (pre-aggregate available)
                pre_aggregate += aggregate;
                break; // Stop looking back
            }
        }

        // Update d_aggregates with pre-aggregate + block_aggregate, set flag to 'P'
        uint64_t block_aggregate = s_data[block_workload - 1];
        uint64_t final_aggregate = pre_aggregate + block_aggregate;
        d_aggregates[blockIdx.x] = (final_aggregate << 2) | 0b10; // Set flag to 'P'
        __threadfence();
    }




    // Share the pre-aggregate within the block
    __shared__ uint64_t shared_pre_aggregate;
    if (tid == 0) {
        shared_pre_aggregate = pre_aggregate;
    }
    __syncthreads();
    pre_aggregate = shared_pre_aggregate;


    // Add pre-aggregate to all elements and write results to global memory
    for (int i = tid; i < block_workload; i += blockDim.x) {
        if (blockStart + i < size) {
            d_output[blockStart + i] = s_data[i] + pre_aggregate;
        }
    }
}





void implementation(const int32_t *d_input, int32_t *d_output, size_t size) {
    const size_t threadsPerBlock = 512;     
    const size_t block_workload = 8160;   
    const size_t blocks = (size + block_workload - 1) / block_workload;

    // Allocate memory for the block aggregates (64-bit for encoding)
    uint64_t *d_aggregates;
    hipMalloc(&d_aggregates, blocks * sizeof(uint64_t));
    hipMemset(d_aggregates, 0, blocks * sizeof(uint64_t)); // Initialize to 'X' (invalid)

    size_t sharedMemorySize = block_workload * sizeof(int32_t);

    koggeStoneDecoupledLookbackKernel<<<blocks, threadsPerBlock, sharedMemorySize>>>(
        d_input, d_output, d_aggregates, size, block_workload);

    hipFree(d_aggregates);
}


